#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "xnor_conv_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__
void binarize_inp_kernel(float* input, int rows, int cols, int channels)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int r = index / rows;
    int c = index % rows;
    if ((r < rows) && (c < cols))
    {
        //printf("%d %d\n",r, c);
        float mean = 0;
        for( int ch = 0; ch < channels; ch++ ) {
            int idx = c + cols * (r + ch * rows);
            mean += input[idx];
        }
        mean /= channels;
        for( int ch = 0; ch < channels; ch++ ) {
            int idx = c + cols * (r + ch * rows);
            input[idx] = input[idx] > 0 ? mean : -mean;
        }
    }
}



void binarize_input_gpu(float *input, int rows, int cols, int channels)
{

    int x = static_cast<int>(ceilf(static_cast<float>(cols) / BLOCK));
    int y = static_cast<int>(ceilf(static_cast<float>(rows) / BLOCK));

    //printf("x: %d x %d %d\n", rows, cols, BLOCK);

    const dim3 grid (x, y, 1);                                // number of blocks
    const dim3 block(BLOCK, 1, 1);

    binarize_inp_kernel<<<cuda_gridsize(rows * cols), BLOCK>>>(input, rows, cols, channels);
    check_error(hipPeekAtLastError());
}


void forward_xnor_conv_layer_gpu(xnor_conv_layer l, network_state state)
{
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = convolutional_out_height(l)*
        convolutional_out_width(l);

    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
    swap_binary(&l);

    for(i = 0; i < l.batch; ++i){

        binarize_input_gpu(state.input + i*l.c*l.h*l.w, l.h, l.w, l.c);

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, l.col_image_gpu);

        float * a = l.filters_gpu;
        float * b = l.col_image_gpu;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }

    if (l.batch_normalize) {
        if (state.train) {
            fast_mean_gpu(l.output_gpu, l.batch, l.n, l.out_h*l.out_w, l.mean_gpu);
            fast_variance_gpu(l.output_gpu, l.mean_gpu, l.batch, l.n, l.out_h*l.out_w, l.variance_gpu);

            scal_ongpu(l.n, .95, l.rolling_mean_gpu, 1);
            axpy_ongpu(l.n, .05, l.mean_gpu, 1, l.rolling_mean_gpu, 1);
            scal_ongpu(l.n, .95, l.rolling_variance_gpu, 1);
            axpy_ongpu(l.n, .05, l.variance_gpu, 1, l.rolling_variance_gpu, 1);

            copy_ongpu(l.outputs*l.batch, l.output_gpu, 1, l.x_gpu, 1);
            normalize_gpu(l.output_gpu, l.mean_gpu, l.variance_gpu, l.batch, l.n, l.out_h*l.out_w);
            copy_ongpu(l.outputs*l.batch, l.output_gpu, 1, l.x_norm_gpu, 1);
        } else {
            normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.n, l.out_h*l.out_w);
        }

        scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.n, l.out_h*l.out_w);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, n);

    activate_array_ongpu(l.output_gpu, m*n*l.batch, l.activation);
    if(l.dot > 0) dot_error_gpu(l);
    swap_binary(&l);
}

void backward_xnor_conv_layer_gpu(xnor_conv_layer l, network_state state)
{
    int i;
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = convolutional_out_height(l)*
        convolutional_out_width(l);

    gradient_array_ongpu(l.output_gpu, m*k*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, k);

    if(l.batch_normalize){
        backward_scale_gpu(l.x_norm_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h, l.scale_updates_gpu);

        scale_bias_gpu(l.delta_gpu, l.scales_gpu, l.batch, l.n, l.out_h*l.out_w);

        fast_mean_delta_gpu(l.delta_gpu, l.variance_gpu, l.batch, l.n, l.out_w*l.out_h, l.mean_delta_gpu);
        fast_variance_delta_gpu(l.x_gpu, l.delta_gpu, l.mean_gpu, l.variance_gpu, l.batch, l.n, l.out_w*l.out_h, l.variance_delta_gpu);
        normalize_delta_gpu(l.x_gpu, l.mean_gpu, l.variance_gpu, l.mean_delta_gpu, l.variance_delta_gpu, l.batch, l.n, l.out_w*l.out_h, l.delta_gpu);
    }

    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = l.col_image_gpu;
        float * c = l.filter_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, l.col_image_gpu);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            swap_binary(&l);
            float * a = l.filters_gpu;
            float * b = l.delta_gpu;
            float * c = l.col_image_gpu;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(l.col_image_gpu, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            swap_binary(&l);
        }
    }
}

void pull_xnor_conv_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void push_xnor_conv_layer(convolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void update_xnor_conv_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);

    axpy_ongpu(size, -decay*batch, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, learning_rate/batch, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, momentum, layer.filter_updates_gpu, 1);
}


